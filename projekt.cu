#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

using namespace std;

#define THREADS 32

__global__ void gray_cuda(const double *rgbpic_flat_r, const double *rgbpic_flat_g, const double *rgbpic_flat_b, double *picture_flat, const int height, const int width){

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    //__syncthreads();
    // int suma1 = 0;
    // int suma2 = 0;
    // int suma3 = 0;
    if (row < height && col < width){   //zle, zrobić osobne macierze dla każdego koloru
        // __syncthreads();
        // suma1 = 0.2627 * rgbpic_flat_r[(row*width) + col];  
        // suma2 = 0.6780 * rgbpic_flat_g[(row*width) + col];
        // suma3 = 0.0593 * rgbpic_flat_b[(row*width) + col];
        picture_flat[(row*width) + col] =  0.2627 * rgbpic_flat_r[(row*width) + col] + 0.6780 * rgbpic_flat_g[(row*width) + col] + 0.0593 * rgbpic_flat_b[(row*width) + col];
    }
}

__global__ void resize_cuda_kernel(const int *picture, int *smolpicture, const int height, const int width, const int xscale, const int yscale, const int thumbheight, const int thumbwidth){

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;
    int amount = xscale * yscale;

    if (row < thumbheight && col < thumbwidth){
        for(int f = 0; f < xscale; ++f){
            for(int g = 0; g < yscale; ++g){
                if(row*(xscale-1) + f < height && col*(yscale-1) + g < width){
                    
                    sum += picture[((row*(xscale-1) + f) * width) + (col*(yscale-1) + g)];
                }
            }
        }
        smolpicture[(row*thumbwidth) + col] = sum/amount;
        sum = 0;
    }
}



int **resize_cuda(int **picture, int height=960, int width=720, int thumbheight=120, int thumbwidth=120){

    // Zadeklarowanie pamięci dla tablicy mniejszego obrazka
    int **smolpicture = new int*[thumbheight];
    for(int i = 0; i<thumbheight;++i) smolpicture[i] = new int[thumbwidth];
    //skala okna do average pooling
    int xscale = 1 + height / thumbheight;
    int yscale = 1 + width / thumbwidth;
    // Obrazek w skali szarości, spłaszczony
    int *picture_flat = new int[height*width];
    int *picture_flat_d = new int[height*width];
    // Pomniejszony obrazek, spłaszczony
    int *smolpicture_flat = new int[thumbheight*thumbwidth];
    int *smolpicture_flat_d = new int[thumbheight*thumbwidth];

    size_t size1 = height*width*sizeof(int);
    size_t size2 = thumbheight*thumbwidth*sizeof(int);

    for(int i = 0; i < height; i++){
        for(int j=0; j < width; j++){
            picture_flat[(i*width) + j] = picture[i][j]; //spłaszczanie obrazka
        }    
    }
    //przygotowanie kernela
    hipMalloc((void **)&picture_flat_d,size1);
    hipMalloc((void **)&smolpicture_flat_d,size2);

    hipMemcpy(picture_flat_d, picture_flat, size1, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(THREADS, THREADS);
    dim3 numBlocks(ceil(thumbheight/float(THREADS)), ceil(thumbwidth/float(THREADS)));

    resize_cuda_kernel<<<numBlocks, threadsPerBlock>>>(picture_flat_d, smolpicture_flat_d, height, width, xscale, yscale, thumbheight, thumbwidth);

    hipMemcpy(smolpicture_flat, smolpicture_flat_d, size2, hipMemcpyDeviceToHost);

    hipFree(smolpicture_flat_d);
    hipFree(picture_flat_d);

    //przepisanie wartości ze spłaszczonej tablicy do tablicy 2D
    for(int i = 0; i < thumbheight; i++){
        for(int j = 0; j < thumbwidth; j++){
            smolpicture[i][j] = smolpicture_flat[(i*thumbwidth) + j];
        }
    }

    //sprzątanie
    delete [] smolpicture_flat;
    delete [] picture_flat;

    hipDeviceReset();

    return smolpicture;
}

int **grayscale_cuda(double **rgbpic, int height=960, int width=720){
    /*
    GraYSCALING NA CUDZIE
    */
    // Zadeklarowanie pamięci na szary obrazek
    int **picture;
    picture = new int* [height];
    for(int i = 0; i<height;++i) picture[i] = new int[width];
    double *picture_flat = new double[height*width];
    double *picture_flat_d = new double[height*width];
    //poszczególne kolory(spektrum RGB)
    double *rgbpic_flat_r = new double[height*width];
    double *rgbpic_flat_g = new double[height*width];
    double *rgbpic_flat_b = new double[height*width];
    //device copies
    double *rgbpic_flat_d_r = new double[height*width];
    double *rgbpic_flat_d_g = new double[height*width];
    double *rgbpic_flat_d_b = new double[height*width];
    
    size_t size2 = height*width*sizeof(double);

    for(int i = 0; i < height; i++){
        for(int j=0; j < 3 * width; j+=3){
            rgbpic_flat_r[(i*width)+ j/3] = rgbpic[i][j]; //"spłaszczanie" tablic
            rgbpic_flat_g[(i*width)+ j/3] = rgbpic[i][j+1]; //"spłaszczanie" tablic
            rgbpic_flat_b[(i*width)+ j/3] = rgbpic[i][j+2]; //"spłaszczanie" tablic     
       }    
    }

    //przygotowanie kernela
    hipMalloc((void **)&picture_flat_d,size2);
    hipMalloc((void **)&rgbpic_flat_d_r,size2);
    hipMalloc((void **)&rgbpic_flat_d_g,size2);
    hipMalloc((void **)&rgbpic_flat_d_b,size2);


    hipMemcpy(rgbpic_flat_d_r, rgbpic_flat_r, size2, hipMemcpyHostToDevice);
    hipMemcpy(rgbpic_flat_d_g, rgbpic_flat_g, size2, hipMemcpyHostToDevice);
    hipMemcpy(rgbpic_flat_d_b, rgbpic_flat_b, size2, hipMemcpyHostToDevice);


    dim3 threadsPerBlock(THREADS, THREADS);
    dim3 numBlocks(ceil(height/float(THREADS)), ceil(width/float(THREADS)));
    gray_cuda<<<numBlocks, threadsPerBlock>>>(rgbpic_flat_d_r, rgbpic_flat_d_g, rgbpic_flat_d_b, picture_flat_d, height, width);

    hipMemcpy(picture_flat, picture_flat_d, size2, hipMemcpyDeviceToHost);

    hipFree(rgbpic_flat_d_r); 
    hipFree(rgbpic_flat_d_g); 
    hipFree(rgbpic_flat_d_b);
    hipFree(picture_flat_d); 

    //przepisanie wartości ze spłaszczonej tablicy do tablicy 2D
    for(int i = 0; i < height; i++){
        for(int j = 0; j < width; j++){
            picture[i][j] = picture_flat[(i*width) + j];
        }
    }

    //sprzątanie
    delete [] picture_flat;
    delete [] rgbpic_flat_r;
    delete [] rgbpic_flat_g;
    delete [] rgbpic_flat_b;

    hipDeviceReset();

    return picture;
}

int main(){
    
    fstream pic;
    fstream wymiary;
    int counteri = 0;
    int counterj = 0;
    // Wczytywać wysokośc i szerokośc z pliku z pythona, thumb też -----------TODO
    int height = 960;
    int width = 720;
    wymiary.open("wymiary.txt", 'r');
    wymiary >> height;
    wymiary >> width;
    wymiary.close();

    double **picture = new double*[height];
    for(int i=0;i<height;i++) picture[i]= new double[3* width]; 
    pic.open("test.txt");
    // zczytanie z pliku wygenerowanego z pythona do tablicy obrazka RGB
    if (pic.is_open()){
        while(!pic.eof()){
            if(counterj<3*width){
                pic >> picture[counteri][counterj];
                pic >> picture[counteri][counterj+1];
                pic >> picture[counteri][counterj+2];
                counterj+=3;
            }
            else{
                counterj = 0;
                counteri++;
            }
        }
    }    
    else{
            cout<<"Error! Nie udało się odczytać pliku!\n";
            return 0;
    }
    pic.close();

    // Otwarcie pliku do zapisu ostatecznego obrazka
    ofstream outcuda;
    outcuda.open("outcuda.txt");
    int **graypicture = new int*[height];
    for(int i =0;i<height;i++) graypicture[i] = new int[width];
    graypicture = grayscale_cuda(picture, height, width); // RGB -> GRAYSCALE

    // To do wczytanie z pliku z pythona
    int thumbheight = 120;
    int thumbwidth = 120;

    int **outpicture = new int*[thumbheight];
    for(int i =0;i<thumbheight;i++) outpicture[i] = new int[thumbwidth];
    
    outpicture = resize_cuda(graypicture, height, width, thumbheight, thumbwidth); //DUŻY -> MNIEJSZY
    for(int i=0;i<thumbheight;i++){
        for(int j=0;j<thumbwidth;j++){
            outcuda << outpicture[i][j] << '\t';
        }
        outcuda << "\n";
    }
    outcuda.close();

    for(int i=0;i<thumbheight;i++) delete[] outpicture[i];
    delete [] outpicture;

    for(int i=0;i<height;i++) delete[] graypicture[i];
    delete [] graypicture;

    return 0;
}

